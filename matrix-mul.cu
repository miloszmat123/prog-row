#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;


void matrixMulCPU(int *c, const int *a, const int *b, int width) {
    for(int y = 0; y < width; ++y) {
        for(int x = 0; x < width; ++x) {
            int sum = 0;
            for(int k = 0; k < width; ++k) {
                sum += a[y * width + k] * b[k * width + x];
            }
            c[y * width + x] = sum;
        }
    }
}

#define BLOCK_SIZE 32

__global__ void matrixMulCUDA(int *c, const int *a, const int *b, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < width && col < width) {
        int temp = 0;
        for(int i = 0; i < width; ++i) {
            temp += a[row * width + i] * b[i * width + col];
        }
        c[row * width + col] = temp;
    }
}

void matrixMul(int* h_a, int* h_b, int* h_c, int width) {
    int size = width * width * sizeof(int);

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, size);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_b, size);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_c, size);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);

    matrixMulCUDA<<<dimGrid, dimBlock>>>(d_c, d_a, d_b, width);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

int main() {
    int WIDTH = 128;
    int size = WIDTH * WIDTH * sizeof(int);
    int* h_a = new int[size];
    int* h_b = new int[size];
    int* h_c_cpu = new int[size];
    int* h_c_gpu = new int[size];

    for(int i = 0; i < WIDTH; i++) {
        for(int j = 0; j < WIDTH; j++) {
            h_a[i * WIDTH + j] = i + j;
            h_b[i * WIDTH + j] = i - j;
        }
    }

    clock_t start = clock();
    matrixMulCPU(h_c_cpu, h_a, h_b, WIDTH);
    clock_t end = clock();

    std::cout << "CPU Matrix Multiplication Time: " << double(end - start) / CLOCKS_PER_SEC << " s\n";

    start = clock();
    matrixMul(h_c_cpu, h_a, h_b, WIDTH);
    end = clock();
    std::cout << "GPU Matrix Multiplication Time: " << double(end - start) / CLOCKS_PER_SEC << " s\n";

    delete[] h_a;
    delete[] h_b;
    delete[] h_c_cpu;
    delete[] h_c_gpu;

    return 0;
}


